#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void MergeSmallBatch_k(float* Input, size_t sizeM, float* Output, int d)
{
	//Example��Block
	int i = threadIdx.x % d; // ith element in d
	int j = (threadIdx.x - i) / d; //jth AB in block
	int k = j + blockIdx.x * (blockDim.x / d); //kth AB in total

	const size_t sizeA = d / 2;
	const size_t sizeB = d / 2;

	Input = Input + k * d;

	float* A = Input;
	float* B = A + sizeA;

	int offset;
	int Kx, Ky;
	int Px, Py;
	int Qx, Qy;

	if (i > sizeA) {
		Kx = i - sizeA;
		Ky = sizeA;
		Px = sizeA;
		Py = i - sizeA;
	}
	else {
		Kx = 0;
		Ky = i;
		Px = i;
		Py = 0;
	}

	while (true) {
		int offset = std::abs(Ky - Py) / 2;
		Qx = Kx + offset;
		Qy = Ky - offset;
		if ((Qy >= 0) && (Qx <= sizeB) &&
			((Qy == sizeA) || (Qx == 0) || (A[Qy] > B[Qx - 1]))) {

			if ((Qx == sizeB) || (Qy == 0) || (A[Qy - 1] <= B[Qx]))
			{
				if ((Qy < sizeA) && ((Qx == sizeB) || (A[Qy] <= B[Qx])))
				{
					Output[i + k * d] = A[Qy];
				}
				else
				{
					Output[i + k * d] = B[Qx];
				}
				break;
			}
			else
			{
				Kx = Qx + 1; Ky = Qy - 1;
			}
		}
		else {
			Px = Qx - 1; Py = Qy + 1;
		}
	}
}

int main() {
	const int Nsamples = 256;
	hipError_t cudaStatus;

	float* InputHost;
	float* OutputHost;
	float* InputCuda;
	float* OutputCuda;

	cudaStatus = hipSetDevice(0);
	for (int d = 2; d <= 4096; d *= 2) {
		hipEvent_t start, end;
		hipEventCreate(&start);
		hipEventCreate(&end);
		InputHost = (float*)malloc(Nsamples * d * sizeof(float));
		OutputHost = (float*)malloc(Nsamples * d * sizeof(float));

		size_t sizeA = d / 2;
		size_t sizeB = d / 2;
		for (int i = 0; i < Nsamples * d; i++) {
			if (i % d == 0 || i % d == sizeA) {
				InputHost[i] = (rand() % 30) * 1.0;
			}
			else {
				InputHost[i] = (rand() % 30) * 1.0 + InputHost[i - 1];
			}
		}

		hipEventRecord(start);
		cudaStatus = hipMalloc((void**)&InputCuda, Nsamples * d * sizeof(float));
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "InputCuda�ڴ����ʧ��");

		cudaStatus = hipMalloc((void**)&OutputCuda, Nsamples * d * sizeof(float));
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "OutputCuda�ڴ����ʧ��");


		cudaStatus = hipMemcpy(InputCuda, InputHost, Nsamples * d * sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "��CPU��������ʧ��!");
		}

		MergeSmallBatch_k << <256, 1024 >> > (InputCuda, Nsamples * d, OutputCuda, d);

		cudaStatus = hipMemcpy(OutputHost, OutputCuda, Nsamples * d * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "��GPU��������ʧ��!");
		}
		hipEventRecord(end);
		hipEventSynchronize(end);

		float t{ 0 };
		hipEventElapsedTime(&t, start, end);

		hipEventDestroy(start);
		hipEventDestroy(end);

		//for (int i = 0; i < 2 * d; ++i) {
		//	std::cout << OutputHost[i] << std::endl;
		//}

		free(InputHost);
		free(OutputHost);
		if (InputCuda != NULL) { hipFree(InputCuda); InputCuda = NULL; }
		if (OutputCuda != NULL) { hipFree(OutputCuda); OutputCuda = NULL; }

		printf("For Length %d, Time has passed %f \n", d, t);
	}


	return 0;
}